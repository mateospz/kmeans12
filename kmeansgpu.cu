#include "hip/hip_runtime.h"
//************************** kmeansgpu.cu ***************************
//*******************Developed by José M. Cecilia*******************
//************************* October 2018************************


#include "hip/hip_runtime.h"
#include "kmeansgpu.h"
#include <hiprand.h>
#include "hip/device_functions.h"
#include <hiprand/hiprand_kernel.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
	if (code != hipSuccess) {
		const char * error = hipGetErrorString(code);
		fprintf(stderr, "GPUassert: %s %s %d\n", error, file, line);
		if (abort) exit(code);
	}
}

/*__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
	
    int index = threadIdx.x;
	hiprand_init(seed, index, 0, &state[index]);
    __syncthreads();
}*/

__global__ void random_init_centroidCUDA(float * cluster_centro_id, float * dataSetMatrix, int clusters, int rows, int columns) {
	
    int tx = threadIdx.x;
    int pos=tx*columns;

//    int random = ceil(hiprand_uniform(&D_state[tx])*rows);
    int random =0;
    for (int i=0; i<columns; i++){
    	cluster_centro_id[pos+i] = dataSetMatrix[random+i];
        //printf ("El random es %f para el thread %d\n", cluster_centro_id[pos+i], tx);
    }   
}



extern "C" int kmeansCUDA(int  dim, float *H_X, float n, int k, float *H_cluster_centroid, int iterations, int *H_cluster_assignment_final) {



    hipDeviceReset();
    return 0;


}






















